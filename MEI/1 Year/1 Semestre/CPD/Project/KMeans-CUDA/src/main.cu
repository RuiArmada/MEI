#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <inttypes.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#define u32 uint32_t

#define cuda_err_check(ans) \
    { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char* file, int line, bool abort = true) {
    if (code != hipSuccess) {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

__device__ float d_euclidean_distance(float x, float y, float x_, float y_) {
    return (x - x_) * (x - x_) + (y - y_) * (y - y_);
}

__global__ void d_clear_accumulators(float* __restrict__ d_xa, float* __restrict__ d_ya, u32* __restrict__ d_ca) {
    u32 i = threadIdx.x;
    d_xa[i] = 0.0f;
    d_ya[i] = 0.0f;
    d_ca[i] = 0;
}

__global__ void d_recalc_centroids(
    float* __restrict__ cx,
    float* __restrict__ cy,
    float* __restrict__ d_xa,
    float* __restrict__ d_ya,
    u32* __restrict__ d_ca) {
    u32 cid = threadIdx.x;
    u32 count = (d_ca[cid] > 0.0f) * d_ca[cid] + (d_ca[cid] <= 0.0f);

    cx[cid] = d_xa[cid] / count;
    cy[cid] = d_ya[cid] / count;
}

// cuda_kMeans_ClearAll_wrapper(
//      outputSums_x_ptr_device,
//      outputSums_y_ptr_device,
//      outputClustersCount_ptr_device,
//      1,
//      CLUSTERS_NUMBER);

__global__ void d_cluster_points(
    float* __restrict__ d_s,
    float* __restrict__ cx,
    float* __restrict__ cy,
    float* __restrict__ d_xa,
    float* __restrict__ d_ya,
    u32* __restrict__ d_ca,
    int n,
    int k) {
    u32 idx = blockIdx.x * blockDim.x + threadIdx.x;
    u32 iter = idx << 1;  // Since we have 2 floats per sample (x, y)

    if (idx < n) {
        float x = d_s[iter];
        float y = d_s[iter + 1];
        float min_dist = 1e10f;
        u32 min_cid = 0;

        for (u32 cid = 0; cid < k; cid++) {
            float x_ = cx[cid];
            float y_ = cy[cid];

            float dist = d_euclidean_distance(x, y, x_, y_);

            if (dist < min_dist) {
                min_dist = dist;
                min_cid = cid;
            }
        }

        // atomic update of the accumulators
        atomicAdd(&d_xa[min_cid], x);
        atomicAdd(&d_ya[min_cid], y);
        atomicAdd(&d_ca[min_cid], 1);
    }
}

void gen_samples(float* s, float* h_cx, float* h_cy, u32 k, u32 n) {
    srand(10);

    // [(x, y), (x, y), ...]
    for (u32 i = 0; i < n; i++) {
        s[i] = (float)rand() / RAND_MAX;
    }

    for (u32 i = 0; i < k; i++) {
        h_cx[i] = s[i * 2];
        h_cy[i] = s[i * 2 + 1];
    }
}

int main(int argc, char** argv) {
    int n = atoi(argv[1]);
    int k = atoi(argv[2]);

    int tpb = 8; // Block Size (Threads Per Block) 128 64 32

    float* h_s = (float*)malloc(n * 2 * sizeof(float));
    float* h_cx = (float*)malloc(k * sizeof(float));
    float* h_cy = (float*)malloc(k * sizeof(float));

    gen_samples(h_s, h_cx, h_cy, k, n * 2);

    // print the first K samples
    //for (int i = 0; i < k; i++)
    //    printf("Centroid %d: (%f, %f)\n", i, h_cx[i], h_cy[i]);

    float* d_s; // samples
    float* d_cx; // centroid x
    float* d_cy; // centroid y
    float* d_xa; // accumulator for x
    float* d_ya; // accumulator for y
    u32* d_ca;  // accumulator for count

    cuda_err_check(hipMalloc(&d_s, n * 2 * sizeof(float)));
    cuda_err_check(hipMalloc(&d_cx, k * sizeof(float)));
    cuda_err_check(hipMalloc(&d_cy, k * sizeof(float)));
    cuda_err_check(hipMalloc(&d_xa, k * sizeof(float)));
    cuda_err_check(hipMalloc(&d_ya, k * sizeof(float)));
    cuda_err_check(hipMalloc(&d_ca, k * sizeof(u32)));

    cuda_err_check(hipMemcpy(d_s, h_s, n * 2 * sizeof(float), hipMemcpyHostToDevice));
    cuda_err_check(hipMemcpy(d_cx, h_cx, k * sizeof(float), hipMemcpyHostToDevice));
    cuda_err_check(hipMemcpy(d_cy, h_cy, k * sizeof(float), hipMemcpyHostToDevice));

    u32 gsz = ceil((float)n / (float)tpb);

    // it's cuda time
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    u32 iter_max = 20;

    for (int iter = 0; iter < iter_max; iter++) {
        d_clear_accumulators << <1, k >> > (d_xa, d_ya, d_ca);
        d_cluster_points << <gsz, tpb >> > (d_s, d_cx, d_cy, d_xa, d_ya, d_ca, n, k);
        hipDeviceSynchronize();
        d_recalc_centroids << <1, k >> > (d_cx, d_cy, d_xa, d_ya, d_ca);
    }

    // allocate memory for the centroids_out
    float* cx_out = (float*)malloc(k * sizeof(float));
    float* cy_out = (float*)malloc(k * sizeof(float));

    cuda_err_check(hipMemcpy(cx_out, d_cx, k * sizeof(float), hipMemcpyDeviceToHost));
    cuda_err_check(hipMemcpy(cy_out, d_cy, k * sizeof(float), hipMemcpyDeviceToHost));

    u32* h_c_acc = (u32*)malloc(k * sizeof(u32));
    cuda_err_check(hipMemcpy(h_c_acc, d_ca, k * sizeof(u32), hipMemcpyDeviceToHost));

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Time taken: %f ms (%f s)\n", milliseconds, milliseconds / 1000.0f);

    // Print the results
    printf("N = %d, K = %d\n", n, k);
    for (int i = 0; i < k; i++)
        printf("Centroid %d: (%f, %f) : Size: %d\n", i, cx_out[i], cy_out[i], h_c_acc[i]);
    printf("Iterations: %d\n", iter_max);

    hipFree(d_s);
    hipFree(d_cx);
    hipFree(d_cy);
    hipFree(d_xa);
    hipFree(d_ya);
    hipFree(d_ca);

    free(h_s);
    free(h_cx);
    free(h_cy);
    free(cx_out);
    free(cy_out);
    free(h_c_acc);
}